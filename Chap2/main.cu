#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloCUDA() {
    printf("Hello CUDA from GPU! thread %d\n", threadIdx.x);
}

int main() {
    printf("Hello GPU from CPU!\n");
    helloCUDA<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
